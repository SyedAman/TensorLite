#include "hip/hip_runtime.h"
#include "TensorLite.h"

// CUDA kernel for forward pass in dense layer
__global__ void denseForwardKernel(float* input, float* weights, float* bias, float* output, int inputSize, int outputSize) {
    // @TODO: CUDA code for matrix multiplication plus bias addition
}

__global__ void denseBackwardKernel(float* gradOutput, float* weights, float* gradInput, int inputSize, int outputSize) {
    // @TODO: CUDA code for backpropagation through dense layer
}

DenseLayer::DenseLayer(int inputSize, int outputSize) : inputSize(inputSize), outputSize(outputSize) {
    // Allocate memory for weights and biases, and initialize them
    weights = new Tensor({inputSize, outputSize});
    bias = new Tensor({outputSize});
    weights->allocateMemoryOnDevice();
    bias->allocateMemoryOnDevice();
}

void DenseLayer::forward(const Tensor &input, Tensor &output) {
    // Use the denseForwardKernel to compute the forward pass
    // Launch the CUDA kernel @TODO: Actual arguments need to be calculated based on block and grid sizes
    denseForwardKernel<<<1, 256>>>(input.device_data, weights->device_data, bias->device_data, output.device_data, inputSize, outputSize);
}

void DenseLayer::backward(const Tensor &input, Tensor &gradInput, const Tensor &gradOutput) {
    // Use the denseBackwardKernel to compute the backward pass and update gradients
    // Launch the CUDA kernel @TODO: Actual arguments need to be calculated based on block and grid sizes
    denseBackwardKernel<<<1, 256>>>(gradOutput.device_data, weights->device_data, gradInput.device_data, inputSize, outputSize);
}
