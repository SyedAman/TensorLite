
#include <hip/hip_runtime.h>
__global__ void vector_add_kernel(float *out, const float *a, const float *b, int n) { // Kernel function
    int index = threadIdx.x + blockIdx.x * blockDim.x; // Calculate the index of the current thread
    if (index < n) { // Ensure the current thread is within the array bounds
        out[index] = a[index] + b[index]; // Perform the vector addition
    }
}

extern "C" void vector_add(float *out, const float *a, const float *b, int n) {
    float *d_a, *d_b,  *d_out;
    size_t size = n * sizeof(float);

    // Allocate memory on the device
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_out, size);

    // Copy inputs to the device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Calculate grid and block sizes
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch the kernel
    vector_add_kernel<<<numBlocks, blockSize>>>(d_out, d_a, d_b, n);

    // Copy result back to host
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}
